#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

 /* This example demonstrates how to use the Cuda OpenGL bindings with the
  * runtime API.
  * Device code.
  */

#ifndef _MAIN_KERNEL_H_
#define _MAIN_KERNEL_H_
#include "cuda_main.h"
#include <cutil_inline.h>
float3* d_binded_fiber;
int binded_fiber_vn=0;
__constant__ float d_subjectMat[16];
__constant__ float d_atlasMat[16];
///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__device__ float3 matrixMultVector(float *m,float3 v)//rv=m*v
{
	float3 rv;
 	//rv.x=m[0]*v.x+m[1]*v.y+m[2]*v.z+m[3];
 	//rv.y=m[4]*v.x+m[5]*v.y+m[6]*v.z+m[7];
 	//rv.z=m[8]*v.x+m[9]*v.y+m[10]*v.z+m[11];
	rv.x=m[0]*v.x+m[4]*v.y+m[8]*v.z+m[12];
	rv.y=m[1]*v.x+m[5]*v.y+m[9]*v.z+m[13];
	rv.z=m[2]*v.x+m[6]*v.y+m[10]*v.z+m[14];
	return rv;
	//rv.w=m[12]*v.x+m[13]*v.y+m[14]*v.z+m[15]*v.w;
}
__global__ void _transformBindedPoints(float3* bindedPoints,int vn, int atlasSizeX, int atlasSizeY,int atlasSizeZ,float3* d_flow_field, int step)
{
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	int nPerAtlasSlice=atlasSizeX*atlasSizeY;
	if(index<vn)
	{
		//the coordinate begin with 1
		//////////////////////////////////////////////////////////////////////////
 	//	float3 current_x1=bindedPoints[index];
		//current_x1=make_float3(xIndex+1,yIndex+1,zIndex+1);
 	//	float3 current_global_coord1=matrixMultVector(d_subjectMat,current_x1);
 	//	float3 current_atlas_coord1=matrixMultVector(d_atlasMat,current_global_coord1);
 	//	d_normMap[index]=current_atlas_coord1;//make_float3(1,1,1);
 	//	return;
		//////////////////////////////////////////////////////////////////////////
		float3 current_x=bindedPoints[index];
		current_x=make_float3(current_x.x+1,current_x.y+1,current_x.z+1);
		float3 current_global_coord=matrixMultVector(d_subjectMat,current_x);
		float3 current_atlas_coord;//=matrixMultVector(d_atlasMat,current_global_coord);
		int3 current_atlas_index;//=make_int3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		float3 current_field;//=d_flow_field[current_atlas_index.z*nPerAtlasSlice+current_atlas_index.y*atlasSizeX+current_atlas_index.x];
		
		for(int i=0;i<step;++i)
		{
			current_atlas_coord=matrixMultVector(d_atlasMat,current_global_coord);
			current_atlas_index=make_int3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
			if(current_atlas_index.x>=atlasSizeX||current_atlas_index.y>=atlasSizeY||current_atlas_index.z>=atlasSizeZ)
			{
				bindedPoints[index]=make_float3(-1,-1,-1);
				return;
			}
			current_field=d_flow_field[current_atlas_index.z*nPerAtlasSlice+current_atlas_index.y*atlasSizeX+current_atlas_index.x];
			current_global_coord=make_float3(current_global_coord.x+current_field.x/step,current_global_coord.y+current_field.y/step,current_global_coord.z+current_field.z/step);			
		}
		current_atlas_coord=matrixMultVector(d_atlasMat,current_global_coord);
		current_atlas_coord=make_float3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		bindedPoints[index]=current_atlas_coord;
	}
}
__global__ void get_Transformaiton_Map(int sizeX,int sizeY,int sizeZ,int atlasSizeX, int atlasSizeY,int atlasSizeZ,float3* d_flow_field,float3* d_normMap, int step)
{
    unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int idx=index;
	int nPerSlice=sizeX*sizeY;
	int zIndex=idx/nPerSlice;
	idx=idx-zIndex*nPerSlice;
	int yIndex=idx/sizeX;
	int xIndex=idx-yIndex*sizeX;
	int nPerAtlasSlice=atlasSizeX*atlasSizeY;
	if(zIndex<sizeZ)
	{
		//the coordinate begin with 1
		//////////////////////////////////////////////////////////////////////////
 		float3 current_x1=make_float3(xIndex+1,yIndex+1,zIndex+1);
 		float3 current_global_coord1=matrixMultVector(d_subjectMat,current_x1);
 		float3 current_atlas_coord1=matrixMultVector(d_atlasMat,current_global_coord1);
 		d_normMap[index]=current_atlas_coord1;//make_float3(1,1,1);

		//////////////////////////////////////////////////////////////////////////
		//float3 current_x=make_float3(xIndex+1,yIndex+1,zIndex+1);
		//float3 current_global_coord=matrixMultVector(d_subjectMat,current_x);
		//float3 current_atlas_coord;//=matrixMultVector(d_atlasMat,current_global_coord);
		//int3 current_atlas_index;//=make_int3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		//float3 current_field;//=d_flow_field[current_atlas_index.z*nPerAtlasSlice+current_atlas_index.y*atlasSizeX+current_atlas_index.x];
		//
		//for(int i=0;i<step;++i)
		//{
		//	current_atlas_coord=matrixMultVector(d_atlasMat,current_global_coord);
		//	current_atlas_index=make_int3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		//	if(current_atlas_index.x>=atlasSizeX||current_atlas_index.y>=atlasSizeY||current_atlas_index.z>=atlasSizeZ)
		//	{
		//		d_normMap[index]=make_float3(-1,-1,-1);
		//		return;
		//	}
		//	current_field=d_flow_field[current_atlas_index.z*nPerAtlasSlice+current_atlas_index.y*atlasSizeX+current_atlas_index.x];
		//	current_global_coord=make_float3(current_global_coord.x+current_field.x/step,current_global_coord.y+current_field.y/step,current_global_coord.z+current_field.z/step);			
		//}
		//current_atlas_coord=matrixMultVector(d_atlasMat,current_global_coord);
		//current_atlas_coord=make_float3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		//d_normMap[index]=current_atlas_coord;
		float3 current_x=make_float3(xIndex+1,yIndex+1,zIndex+1);
		float3 current_global_coord=matrixMultVector(d_subjectMat,current_x);
		float3 current_atlas_coord;//=matrixMultVector(d_atlasMat,current_global_coord);
		int3 current_atlas_index;//=make_int3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		float3 current_field;//=d_flow_field[current_atlas_index.z*nPerAtlasSlice+current_atlas_index.y*atlasSizeX+current_atlas_index.x];
		
		for(int i=0;i<step;++i)
		{
			current_atlas_coord=matrixMultVector(d_atlasMat,current_global_coord);
			current_atlas_index=make_int3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
			if(current_atlas_index.x>=atlasSizeX||current_atlas_index.y>=atlasSizeY||current_atlas_index.z>=atlasSizeZ)
			{
				d_normMap[index]=make_float3(-1,-1,-1);
				return;
			}
			current_field=d_flow_field[current_atlas_index.z*nPerAtlasSlice+current_atlas_index.y*atlasSizeX+current_atlas_index.x];
			current_global_coord=make_float3(current_global_coord.x+current_field.x/step,current_global_coord.y+current_field.y/step,current_global_coord.z+current_field.z/step);			
		}
		current_atlas_coord=matrixMultVector(d_atlasMat,current_global_coord);
		current_atlas_coord=make_float3(current_atlas_coord.x-1,current_atlas_coord.y-1,current_atlas_coord.z-1);
		d_normMap[index].y=current_atlas_coord.x;
	}
}
__global__ void get_matrix(float3* f1, float3* f2, int n1, int n2,float* resultMatrix,int pitch)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;
	if(idx<n1&&idy<n2)
	{
		float3 distXYZ=make_float3(f1[idx].x-f2[idy].x,f1[idx].y-f2[idy].y,f1[idx].z-f2[idy].z);
		float dist=distXYZ.x*distXYZ.x+distXYZ.y*distXYZ.y+distXYZ.z*distXYZ.z;
		float* row = (float*)((char*)resultMatrix + idy * pitch);
		row[idx] = dist; 
	}
}
__global__ void get_max(float* input_Matrix,int input_pitch,int input_w, int input_h,float* result_Matrix,int result_pitch,int result_w, int result_h)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;	
	if(idx<result_w&&idy<result_h)
	{
		float* input_row = (float*)((char*)input_Matrix + idy * input_pitch);
		float* result_row = (float*)((char*)result_Matrix + idy * result_pitch);
		int src1=idx<<1;
		int src2=src1+1;
		float result=input_row[src1];
		if(src2<input_w)
		{
			float result2=input_row[src2];
			if(result2>result)result=result2;
		}
		result_row[idx] = result; 
	}
}
//shrink column
__global__ void get_min_array(float* input_Matrix,int input_pitch,int input_w, int input_h,float* result_Matrix,int result_pitch, int result_h)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y*blockDim.y + threadIdx.y;	
	if(idx<input_w&&idy<result_h)
	{
		int src1=idy<<1;
		int src2=src1+1;
		float* input_row1 = (float*)((char*)input_Matrix + src1 * input_pitch);
		float* input_row2 = (float*)((char*)input_Matrix + src2 * input_pitch);
		
		float* result_row = (float*)((char*)result_Matrix + idy * result_pitch);
		float result=input_row1[idx];
		if(src2<input_h)
		{
			float result2=input_row2[idx];
			if(result2<result)result=result2;
		}
		result_row[idx] = result; 
	}
}
__global__ void get_min(float* input_Matrix,int input_pitch, int input_h,float* result_Matrix,int result_pitch, int result_h)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;	
	if(idx<result_h)
	{
		int src1=idx<<1;
		int src2=src1+1;
		float* input_row1 = (float*)((char*)input_Matrix + src1 * input_pitch);
		float* input_row2 = (float*)((char*)input_Matrix + src2 * input_pitch);
		float* result_row = (float*)((char*)result_Matrix + idx * result_pitch);
		float result=input_row1[0];
		if(src2<input_h)
		{
			float result2=input_row2[0];
			if(result2<result)result=result2;
		}
		result_row[0] = result; 
	}
}
void get_min_array(float3* fiber, int vn,float * min_array)
{
    // execute the kernel
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	//test
//     float* resultMatrix[2];
// 	resultMatrix[0]=new float[vn*binded_fiber_vn];
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	dim3 dimBlock(CUDA_BlockSize_X,CUDA_BlockSize_Y);
	dim3 dimGrid((binded_fiber_vn + dimBlock.x - 1)/dimBlock.x,(vn + dimBlock.y - 1)/dimBlock.y);
	float* d_resultMatrix[2];
	int pitch[2];

	float3* d_fiber;
	CUDA_SAFE_CALL( hipMalloc((void **) &d_fiber, vn * sizeof(float3)));
	CUDA_SAFE_CALL( hipMemcpy(d_fiber,fiber, vn * sizeof(float3), hipMemcpyHostToDevice) ); 	
	int input_h=vn;	
	int result_h=(vn+1)/2;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&d_resultMatrix[0],(size_t  *) &pitch[0], (size_t)binded_fiber_vn * sizeof(float),(size_t) vn));
	
	get_matrix<<< dimGrid, dimBlock >>>(d_binded_fiber,d_fiber,binded_fiber_vn,vn,d_resultMatrix[0],pitch[0]);
	CUDA_SAFE_CALL(hipFree(d_fiber));

	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	//test
// 	cutilSafeCall( hipMemcpy2D( resultMatrix[0], binded_fiber_vn*sizeof(float), d_resultMatrix[0],pitch[0],sizeof(float)*binded_fiber_vn,vn, hipMemcpyDeviceToHost) ); 
// 	for(int j=0;j<vn;++j)
// 	{
// 		for(int i=0;i<binded_fiber_vn;++i)
// 		{
// 			printf("%f\t",resultMatrix[0][j*binded_fiber_vn+i]);
// 		}
// 		printf("\n");
// 	}
// 	resultMatrix[1]=new float[result_h*binded_fiber_vn];
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	CUDA_SAFE_CALL(hipMallocPitch((void**)&d_resultMatrix[1],(size_t  *) &pitch[1], (size_t)binded_fiber_vn * sizeof(float),(size_t) result_h));
	int current_result_buffer=1;
	int current_input_buffer=0;
	while(input_h>1)
	{
		dimGrid.y=(result_h + dimBlock.y - 1)/dimBlock.y;
		get_min_array<<< dimGrid, dimBlock >>>( d_resultMatrix[current_input_buffer],pitch[current_input_buffer],binded_fiber_vn, input_h,d_resultMatrix[current_result_buffer],pitch[current_result_buffer], result_h);
		input_h=result_h;
		result_h=(result_h+1)/2;
		current_result_buffer=current_input_buffer;
		current_input_buffer=1-current_input_buffer;
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		//test
// 		cutilSafeCall( hipMemcpy2D( resultMatrix[current_input_buffer], binded_fiber_vn*sizeof(float), d_resultMatrix[current_input_buffer],pitch[current_input_buffer],sizeof(float)*binded_fiber_vn,input_h, hipMemcpyDeviceToHost) ); 
// 		//cutilSafeCall( hipMemcpy2D( resultMatrix[0], n1*sizeof(float), d_resultMatrix[0],pitch[0],sizeof(float)*n1,n2, hipMemcpyDeviceToHost) ); 
// 		//cutilSafeCall( hipMemcpy2D( resultMatrix[1], n12*sizeof(float), d_resultMatrix[1],pitch[1],n12*sizeof(float),n2, hipMemcpyDeviceToHost) ); 
// 		printf("\n");
// 		for(int j=0;j<input_h;++j)
// 		{
// 			for(int i=0;i<binded_fiber_vn;++i)
// 			{
// 				printf("%f\t",resultMatrix[current_input_buffer][j*binded_fiber_vn+i]);
// // 				if(current_input_buffer==0)
// // 					printf("%f\t",resultMatrix[0][j*n1+i]);
// // 				else
// // 					printf("%f\t",resultMatrix[1][j*n12+i]);
// 			}
// 			printf("\n");
// 		}
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
	}
	cutilSafeCall( hipMemcpy2D( min_array, binded_fiber_vn*sizeof(float), d_resultMatrix[current_input_buffer],pitch[current_input_buffer],sizeof(float)*binded_fiber_vn,1, hipMemcpyDeviceToHost) ); 
	CUDA_SAFE_CALL(hipFree(d_resultMatrix[0]));
	CUDA_SAFE_CALL(hipFree(d_resultMatrix[1]));
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	//test
// 	delete [] resultMatrix[0];
// 	delete [] resultMatrix[1];
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	
}
float fiber_similarity_(float3* f1, float3* f2, int n1, int n2)
{
    // execute the kernel
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	//test
//     float* resultMatrix[2];
// 	resultMatrix[0]=new float[n1*n2];
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	dim3 dimBlock(CUDA_BlockSize_X,CUDA_BlockSize_Y);
	dim3 dimGrid((n1 + dimBlock.x - 1)/dimBlock.x,(n2 + dimBlock.y - 1)/dimBlock.y);
	float* d_resultMatrix[2];
	int pitch[2];
	float3* d_f1;
	float3* d_f2;
	CUDA_SAFE_CALL( hipMalloc((void **) &d_f1, n1 * sizeof(float3))); 
	CUDA_SAFE_CALL( hipMalloc((void **) &d_f2, n2 * sizeof(float3)));
	CUDA_SAFE_CALL( hipMemcpy(d_f1,f1, n1 * sizeof(float3), hipMemcpyHostToDevice) ); 	
	CUDA_SAFE_CALL( hipMemcpy(d_f2,f2, n2 * sizeof(float3), hipMemcpyHostToDevice) ); 	
	int input_w=n1;
	
	int result_w=(n1+1)/2;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&d_resultMatrix[0],(size_t  *) &pitch[0], (size_t)n1 * sizeof(float),(size_t) n2));
	
	get_matrix<<< dimGrid, dimBlock >>>(d_f1,d_f2,n1,n2,d_resultMatrix[0],pitch[0]);
	CUDA_SAFE_CALL(hipFree(d_f1));
	CUDA_SAFE_CALL(hipFree(d_f2));
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	//test
// 	cutilSafeCall( hipMemcpy2D( resultMatrix[0], n1*sizeof(float), d_resultMatrix[0],pitch[0],sizeof(float)*n1,n2, hipMemcpyDeviceToHost) ); 
// 	for(int j=0;j<n2;++j)
// 	{
// 		for(int i=0;i<n1;++i)
// 		{
// 			printf("%f\t",resultMatrix[0][j*n1+i]);
// 		}
// 		printf("\n");
// 	}
// 	resultMatrix[1]=new float[result_w*n2];
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	CUDA_SAFE_CALL(hipMallocPitch((void**)&d_resultMatrix[1],(size_t  *) &pitch[1], result_w * sizeof(float),(size_t) n2));
	int current_result_buffer=1;
	int current_input_buffer=0;
	while(input_w>1)
	{
		dimGrid.x=(result_w + dimBlock.x - 1)/dimBlock.x;
		get_max<<< dimGrid, dimBlock >>>( d_resultMatrix[current_input_buffer],pitch[current_input_buffer],input_w, n2,d_resultMatrix[current_result_buffer],pitch[current_result_buffer],result_w, n2);
		input_w=result_w;
		result_w=(result_w+1)/2;
		current_result_buffer=current_input_buffer;
		current_input_buffer=1-current_input_buffer;
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		//test
// 		cutilSafeCall( hipMemcpy2D( resultMatrix[current_input_buffer], input_w*sizeof(float), d_resultMatrix[current_input_buffer],pitch[current_input_buffer],sizeof(float)*input_w,n2, hipMemcpyDeviceToHost) ); 
// 		//cutilSafeCall( hipMemcpy2D( resultMatrix[0], n1*sizeof(float), d_resultMatrix[0],pitch[0],sizeof(float)*n1,n2, hipMemcpyDeviceToHost) ); 
// 		//cutilSafeCall( hipMemcpy2D( resultMatrix[1], n12*sizeof(float), d_resultMatrix[1],pitch[1],n12*sizeof(float),n2, hipMemcpyDeviceToHost) ); 
// 		printf("\n");
// 		for(int j=0;j<n2;++j)
// 		{
// 			for(int i=0;i<input_w;++i)
// 			{
// 				printf("%f\t",resultMatrix[current_input_buffer][j*input_w+i]);
// // 				if(current_input_buffer==0)
// // 					printf("%f\t",resultMatrix[0][j*n1+i]);
// // 				else
// // 					printf("%f\t",resultMatrix[1][j*n12+i]);
// 			}
// 			printf("\n");
// 		}
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
	}
	int threadsPerBlock = CUDA_BlockSize;
	int blocksPerGrid;// = (n2 + threadsPerBlock - 1) / threadsPerBlock;
	int input_h=n2;
	int result_h=(n2+1)/2;
	while(input_h>1)
	{		
		blocksPerGrid = (result_h + threadsPerBlock - 1) / threadsPerBlock;
		get_min<<<blocksPerGrid, threadsPerBlock>>>(d_resultMatrix[current_input_buffer],pitch[current_input_buffer], input_h,d_resultMatrix[current_result_buffer],pitch[current_result_buffer], result_h);
		input_h=result_h;
		result_h=(result_h+1)/2;
		current_result_buffer=current_input_buffer;
		current_input_buffer=1-current_input_buffer;
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
		//test
// 		cutilSafeCall( hipMemcpy2D( resultMatrix[current_input_buffer], sizeof(float), d_resultMatrix[current_input_buffer],pitch[current_input_buffer],sizeof(float),input_h, hipMemcpyDeviceToHost) ); 
// 		//cutilSafeCall( hipMemcpy2D( resultMatrix[0], n1*sizeof(float), d_resultMatrix[0],pitch[0],sizeof(float)*n1,n2, hipMemcpyDeviceToHost) ); 
// 		//cutilSafeCall( hipMemcpy2D( resultMatrix[1], n12*sizeof(float), d_resultMatrix[1],pitch[1],n12*sizeof(float),n2, hipMemcpyDeviceToHost) ); 
// 		printf("\n");
// 		for(int j=0;j<input_h;++j)
// 		{
// 			printf("%f\t",resultMatrix[current_input_buffer][j]);
// // 			if(current_input_buffer==0)
// // 				printf("%f\t",resultMatrix[0][j*n1]);
// // 			else
// // 				printf("%f\t",resultMatrix[1][j*n12]);
// 			printf("\n");
// 		}
		///////////////////////////////////////////////////////////////////////////////////////////////////////////
	}
	float result_;
	cutilSafeCall( hipMemcpy2D( &result_, sizeof(float), d_resultMatrix[current_input_buffer],pitch[current_input_buffer],sizeof(float),1, hipMemcpyDeviceToHost) ); 
	CUDA_SAFE_CALL(hipFree(d_resultMatrix[0]));
	CUDA_SAFE_CALL(hipFree(d_resultMatrix[1]));
	
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	//test
// 	delete [] resultMatrix[0];
// 	delete [] resultMatrix[1];
	///////////////////////////////////////////////////////////////////////////////////////////////////////////
	return result_;
	
}
// Wrapper for the __global__ call that sets up the kernel call
float fiber_similarity(float* f1, float* f2, int n1, int n2)
{
	float3* f1_3=(float3*)f1;
	float3* f2_3=(float3*)f2;
	return fiber_similarity_(f1_3,f2_3,  n1,  n2);
}
void bind_fiber(float* fiber_data,int vn)
{
	if(binded_fiber_vn!=0)
		unbind_fiber();
	float3* binded_fiber=(float3*)fiber_data;
	binded_fiber_vn=vn;
	CUDA_SAFE_CALL( hipMalloc((void **) &d_binded_fiber, vn * sizeof(float3))); 
	CUDA_SAFE_CALL( hipMemcpy(d_binded_fiber,binded_fiber, vn * sizeof(float3), hipMemcpyHostToDevice) ); 		
}
void unbind_fiber()
{
	CUDA_SAFE_CALL(hipFree(d_binded_fiber));
	binded_fiber_vn=0;
}
void similarity_with_binded_fiber(float* fiber, int vn,float * min_array)
{
	float3* fiber_3=(float3*)fiber;
	get_min_array(fiber_3, vn,min_array);
}
void getTransformaitonMap(float* subjectMat,int sizeX,int sizeY,int sizeZ,float* atlasMat, int atlasSizeX, int atlasSizeY, int atlasSizeZ,float* flow_field,float* _normMap)
{
	unsigned int subject_size=sizeX*sizeY*sizeZ;
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("d_subjectMat"), (const void *)subjectMat, sizeof(float)*16, 0, hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("d_atlasMat"), (const void *)atlasMat, sizeof(float)*16, 0, hipMemcpyHostToDevice));
	float3* d_normMap;
	CUDA_SAFE_CALL( hipMalloc((void **) &d_normMap, subject_size * sizeof(float3))); 
	float3* d_flow_field;
	CUDA_SAFE_CALL( hipMalloc((void **) &d_flow_field, atlasSizeX*atlasSizeY*atlasSizeZ * sizeof(float3))); 
	CUDA_SAFE_CALL( hipMemcpy(d_flow_field,(float3*)flow_field, atlasSizeX*atlasSizeY*atlasSizeZ * sizeof(float3), hipMemcpyHostToDevice) ); 
	dim3 dimBlock(CUDA_BlockSize);
	dim3 dimGrid(( subject_size+ dimBlock.x - 1)/dimBlock.x);
	get_Transformaiton_Map<<< dimGrid, dimBlock >>>(sizeX,sizeY,sizeZ,atlasSizeX, atlasSizeY,atlasSizeZ,d_flow_field, d_normMap,  64);
	CUDA_SAFE_CALL( hipMemcpy((float3*)_normMap,d_normMap, subject_size * sizeof(float3), hipMemcpyDeviceToHost) ); 
	////////////////////////////////////////////////////////////////////////////
// 	float3* debugmap=(float3*)_normMap;
// 	int index=0;
// 	for(int k=0;k<sizeZ;k++)
// 	{
// 		for(int j=0;j<sizeY;j++)
// 		{
// 			for(int i=0;i<sizeX;i++)
// 			{
// 				if(debugmap[index].x>atlasSizeX||debugmap[index].y>atlasSizeY||debugmap[index].z>atlasSizeZ)
// 				{
// 					printf("error");
// 				}
// 				++index;
// 			}
// 		}
// 	}
	////////////////////////////////////////////////////////////////////////////
	CUDA_SAFE_CALL(hipFree(d_normMap));
	CUDA_SAFE_CALL(hipFree(d_flow_field));
		
}
 void transformBindedPoints(float* fiberPoints, int vn, float* subjectMat,float* atlasMat, int atlasSizeX, int atlasSizeY, int atlasSizeZ,float* flow_field)
 {
	//if(binded_fiber_vn!=vn)
	{
		bind_fiber(fiberPoints,vn);
	}
 	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("d_subjectMat"), (const void *)subjectMat, sizeof(float)*16, 0, hipMemcpyHostToDevice));
 	cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL("d_atlasMat"), (const void *)atlasMat, sizeof(float)*16, 0, hipMemcpyHostToDevice));
 	float3* d_flow_field;
 	CUDA_SAFE_CALL( hipMalloc((void **) &d_flow_field, atlasSizeX*atlasSizeY*atlasSizeZ * sizeof(float3))); 
 	CUDA_SAFE_CALL( hipMemcpy(d_flow_field,(float3*)flow_field, atlasSizeX*atlasSizeY*atlasSizeZ * sizeof(float3), hipMemcpyHostToDevice) ); 
 	dim3 dimBlock(CUDA_BlockSize);
 	dim3 dimGrid(( binded_fiber_vn+ dimBlock.x - 1)/dimBlock.x);
	_transformBindedPoints<<< dimGrid, dimBlock >>>(d_binded_fiber,binded_fiber_vn,atlasSizeX, atlasSizeY,atlasSizeZ,d_flow_field,  64);
 	CUDA_SAFE_CALL( hipMemcpy((float3*)fiberPoints,d_binded_fiber, vn * sizeof(float3), hipMemcpyDeviceToHost) ); 
 	////////////////////////////////////////////////////////////////////////////
 // 	float3* debugmap=(float3*)_normMap;
 // 	int index=0;
 // 	for(int k=0;k<sizeZ;k++)
 // 	{
 // 		for(int j=0;j<sizeY;j++)
 // 		{
 // 			for(int i=0;i<sizeX;i++)
 // 			{
 // 				if(debugmap[index].x>atlasSizeX||debugmap[index].y>atlasSizeY||debugmap[index].z>atlasSizeZ)
 // 				{
 // 					printf("error");
 // 				}
 // 				++index;
 // 			}
 // 		}
 // 	}
 	////////////////////////////////////////////////////////////////////////////
 	CUDA_SAFE_CALL(hipFree(d_flow_field));
 		
 }
#endif // #ifndef _SIMPLEGL_KERNEL_H_
